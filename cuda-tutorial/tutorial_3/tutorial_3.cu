

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 50000


const int threads_per_block = 256;

__global__
void dot_gpu(float *a, float *b, float *c) {
    __shared__
        float cache[threads_per_block];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIndex] = temp;

}

int main(int argc, char *argv[]) {

    std::clock_t start_t;
    double duration;
    start_t = std::clock();
    
    int a[N];
    int b[N];
    int c[N];

    int *dev_a;
    int *dev_b;
    int *dev_c;

    // gpu timer
    hipEvent_t start;
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    
    // allocate memory on GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));
    
    for (int i = 0; i < N; i++) {
        a[i] = -1;
        b[i] = i * i;
    }

    // copy 2 arrays to device memory
    hipMemcpy(dev_a, a, N * sizeof(N), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(N), hipMemcpyHostToDevice);


    // copy from device to host
    hipMemcpy(c, dev_c, N * sizeof(N), hipMemcpyDeviceToHost);

    // for (int i = 0; i < N; i++) {
    //     std::cout << a[i] << " + " << b[i] << " = " << c[i] << "\n";
    // }

    // add_cpu(a, b, c);

    hipEvent_t stop;
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float dev_time;
    hipEventElapsedTime(&dev_time, start, stop);
    std::cout << "Time: " << dev_time << "\n";
    
    std::cout << "DONE" << "\n";
    duration = ( std::clock() - start_t ) / (double) CLOCKS_PER_SEC;
    std::cout<<"printf: "<< duration <<'\n';
    
    
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
    return 0;
}
