

#include <hip/hip_runtime.h>
#include <iostream>

// inform that the function should run on device instead of the host
__global__
void add_kernel(int a, int b, int *c) {
    *c = a + b;
}

int main(int argc, char *argv[]) {

    // get device info
    int count;
    hipGetDeviceCount(&count);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::cout << "Device Count: " << count << "\n";
    std::cout << "Name: " << prop.name  << "\n";
    std::cout << "\t mem: " << prop.maxThreadsPerBlock  << "\n";
    
    
    int c;
    int *dev_c;
    
    // allocate memory on the device
    // returned pointer should not be dereferenced
    hipMalloc((void**)&dev_c, sizeof(int));
    
    // used to send device code to device compiler
    // angle brackets denote arguments we plan to pass for the device
    add_kernel<<< 1, 1 >>> (2, 7, dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "SUM: " << c  << "\n";

    hipFree(dev_c);
    
    std::cout << "HELLO WORLD!"  << "\n";
    return 0;
}
